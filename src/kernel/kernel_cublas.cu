#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "gpublas.h"
#include "kernel_cublas.h"

#ifdef CUBLAS_WO_DT
void kernel_rocblasDgemm (hipblasHandle_t handle, unsigned int m, unsigned int k, const double* a, const double* b, double* c)
{
    const double alpha = 1.0f; 
    const double beta = 0.0f;

    // C[mxm] = A[mxk] * B[kxm]
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, m, k, &alpha, a, m, b, k, &beta, c, m);

}

void kernel_rocblasSgemm (hipblasHandle_t handle, unsigned int m, unsigned int k, const float* a, const float* b, float* c)
{
    const float alpha = 1.0f; 
    const float beta = 0.0f;

    // C[mxm] = A[mxk] * B[kxm]
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, m, k, &alpha, a, m, b, k, &beta, c, m);

}
#endif
#ifdef CUBLAS
void kernel_cublasDgemm (hipblasHandle_t handle, unsigned int m, unsigned int k, const double* a, const double* b, double* c)
{
    int size_ab = m * k * sizeof(double);
    int size_c  = m * m * sizeof(double);
    
    double* d_a;
    double* d_b;
    double* d_c;

	CHECK(hipMalloc(&d_a, size_ab));
    CHECK(hipMalloc(&d_b, size_ab));
    CHECK(hipMalloc(&d_c, size_c));

    CHECK(hipMemcpy(d_a, a, size_ab, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_b, b, size_ab, hipMemcpyHostToDevice));

    double alpha = 1.0f;
    double beta = 0.0f;
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, m, k, &alpha, d_a, m, d_b, k, &beta, d_c, m);
        
	CHECK(hipMemcpy(c, d_c, size_c, hipMemcpyDeviceToHost));

    CHECK(hipFree(d_a));
    CHECK(hipFree(d_b));
    CHECK(hipFree(d_c));
}

void kernel_cublasSgemm (hipblasHandle_t handle, unsigned int m, unsigned int k, const float* a, const float* b, float* c)
{
    int size_ab = m * k * sizeof(float);
    int size_c  = m * m * sizeof(float);
    
    float* d_a;
    float* d_b;
    float* d_c;

	CHECK(hipMalloc(&d_a, size_ab));
    CHECK(hipMalloc(&d_b, size_ab));
    CHECK(hipMalloc(&d_c, size_c));

    CHECK(hipMemcpy(d_a, a, size_ab, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_b, b, size_ab, hipMemcpyHostToDevice));

    float alpha = 1.0f;
    float beta = 0.0f;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, m, k, &alpha, d_a, m, d_b, k, &beta, d_c, m);
        
	CHECK(hipMemcpy(c, d_c, size_c, hipMemcpyDeviceToHost));

    CHECK(hipFree(d_a));
    CHECK(hipFree(d_b));
    CHECK(hipFree(d_c));
}
#endif