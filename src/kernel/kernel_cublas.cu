#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "kernel_cublas.h"

void kernel_cublasDgemm (hipblasHandle_t handle, unsigned int m, unsigned int k, const double* a, const double* b, double* c)
{
    int size_ab = m * k * sizeof(double);
    int size_c  = m * m * sizeof(double);
    
    double* d_a;
    double* d_b;
    double* d_c;

	CHECK(hipMalloc(&d_a, size_ab));
    CHECK(hipMalloc(&d_b, size_ab));
    CHECK(hipMalloc(&d_c, size_c));

    CHECK(hipMemcpy(d_a, a, size_ab, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_b, b, size_ab, hipMemcpyHostToDevice));

    double alpha = 1.0f;
    double beta = 0.0f;
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, m, k, &alpha, d_a, m, d_b, k, &beta, d_c, m);
        
	CHECK(hipMemcpy(c, d_c, size_c, hipMemcpyDeviceToHost));

    CHECK(hipFree(d_a));
    CHECK(hipFree(d_b));
    CHECK(hipFree(d_c));
}

void kernel_cublasSgemm (hipblasHandle_t handle, unsigned int m, unsigned int k, const float* a, const float* b, float* c)
{
    int size_ab = m * k * sizeof(float);
    int size_c  = m * m * sizeof(float);
    
    float* d_a;
    float* d_b;
    float* d_c;

	CHECK(hipMalloc(&d_a, size_ab));
    CHECK(hipMalloc(&d_b, size_ab));
    CHECK(hipMalloc(&d_c, size_c));

    CHECK(hipMemcpy(d_a, a, size_ab, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_b, b, size_ab, hipMemcpyHostToDevice));

    float alpha = 1.0f;
    float beta = 0.0f;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, m, k, &alpha, d_a, m, d_b, k, &beta, d_c, m);
        
	CHECK(hipMemcpy(c, d_c, size_c, hipMemcpyDeviceToHost));

    CHECK(hipFree(d_a));
    CHECK(hipFree(d_b));
    CHECK(hipFree(d_c));
}